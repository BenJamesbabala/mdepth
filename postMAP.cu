#include "hip/hip_runtime.h"
/* 
--Ayan Chakrabarti <ayanc@ttic.edu>
*/

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <stdint.h>

#define F float

#define NUMT 1024

void __global__ postMAP(F * der, F * pred, F * bins, F beta,
			int W, int H, int K, int B, int crop) {


  int i,j,x,y,k,W2,H2;
  F btp1,brat, dmin, cmin, dj, cj, dcur;

  btp1 = 1.0 + beta; brat = beta / btp1;
  W2 = W + 2*crop;
  H2 = H + 2*crop;
  
  for (i = blockIdx.x * blockDim.x + threadIdx.x;
       i < W*H*K;
       i += blockDim.x * gridDim.x) {

    k = i/(W*H); x = i%(W*H); y = x%H; x = x/H;

    cmin = pred[y+x*H+k*W*H];
    dmin = bins[k]; 

    if(beta > 0.0) {
      dcur = der[(y+crop)+(x+crop)*H2+k*W2*H2];
      cmin = cmin + brat*(dmin-dcur)*(dmin-dcur);
      dmin = (dmin + beta*dcur) / btp1;
    }

    for(j = 1; j < B; j++) {
      cj = pred[y+x*H+k*W*H+j*W*H*K];
      dj = bins[k+j*K];

      if(beta > 0.0) {
	cj = cj + brat*(dj-dcur)*(dj-dcur);
	dj = (dj + beta*dcur) / btp1;
      }

      if(cj < cmin) {cmin = cj; dmin = dj;};
    }

    der[(y+crop)+(x+crop)*H2+k*W2*H2] = dmin;
  }

}


F * getGPUmem(const char * name) {

  const mxGPUArray * tmp;
  F * dptr;

  if(!mxIsGPUArray(mexGetVariablePtr("caller",name)))
    mexPrintf("%s is not on gpu!\n",name);

  tmp = mxGPUCreateFromMxArray(mexGetVariablePtr("caller",name));
  dptr = (F*) mxGPUGetDataReadOnly(tmp);
  mxGPUDestroyGPUArray(tmp);

  return (F*) dptr;
}

/*
  function postMAP(beta,crop,H,W,K,B)
       X, pred, bins need to be present in the caller workspace.
*/
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {

  F * der, * pred, * bins, beta;
  int crop,H,W,K,B;

  beta = mxGetScalar(prhs[0]);
  crop = (int) mxGetScalar(prhs[1]);
  H = (int) mxGetScalar(prhs[2]);
  W = (int) mxGetScalar(prhs[3]);
  K = (int) mxGetScalar(prhs[4]);
  B = (int) mxGetScalar(prhs[5]);

  
  der = getGPUmem("X"); pred = getGPUmem("pred"); bins = getGPUmem("bins");
  postMAP<<<(W*H*K+NUMT-1)/NUMT,NUMT>>>(der,pred,bins,beta,W,H,K,B,crop);
}
